#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void gpuAdd(int *d_a, int *d_b, int* d_c)
{
	*d_c = *d_a + *d_b;
}
int main(void)
{
	int h_a, h_b, h_c;//host var to store answer
	int *d_a, *d_b, *d_c;//device pointer to store answer
	hipMalloc((void**)&d_a, sizeof(int));
	hipMalloc((void**)&d_b, sizeof(int));
	hipMalloc((void**)&d_c, sizeof(int));

	std::cout << "enter 2 integers?";
	std::cin >> h_a >> h_b;
	//copy host to device
	hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &h_b, , hipMemcpyHostToDevice);
	
	gpuAdd << <1, 1>> > (d_a, d_b, d_c);
	//copy results from device to host
	hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);

	printf("%d + %d = %d\n", h_a, h_b, h_c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
