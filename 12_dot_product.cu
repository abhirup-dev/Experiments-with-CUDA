#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include <cmath>

#define N 1024
#define threads_per_block 512

template<typename T>
__global__ void blockwise_dot(T *d_a, T *d_b, T *block_sum)
{
	__shared__ T partial_sum [threads_per_block];
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int idx = threadIdx.x;
	T sum=0;

	while(tid < N)
	{
		sum += d_a[tid]*d_b[tid];
		tid += blockDim.x * gridDim.x;
	}

	//store partial sum of threads of current block
	//in quickly accessible shared memory
	partial_sum[idx] = sum;

	//sync all threads
	__syncthreads();

	int i = blockDim.x /2;
	while(i != 0)
	{
		if(idx < i)
			partial_sum[idx] += partial_sum[idx+i];
		__syncthreads();
		i /= 2;
	}

	if(idx == 0)
		block_sum[blockIdx.x] = partial_sum[0];

}

int main()
{
	int num_blocks = std::ceil(float(N)/threads_per_block);
	float h_a[N], h_b[N], *d_a, *d_b, *d_partsum;
	for(int i=0; i<N; i++)
	{
		h_a[i] = i; h_b[i] = 1;
	}

	printf("#blocks %d #threads/block %d\n", num_blocks, threads_per_block);

	hipMalloc((void**)&d_a, N*sizeof(float));
	hipMalloc((void**)&d_b, N*sizeof(float));
	hipMalloc((void**)&d_partsum, num_blocks*sizeof(float));

	hipMemcpy(d_a, &h_a, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &h_b, N*sizeof(float), hipMemcpyHostToDevice);

	blockwise_dot<float> <<<num_blocks,threads_per_block>>> (d_a, d_b, d_partsum);

	hipDeviceSynchronize();

	float h_partsum[num_blocks], total_sum=0;
	hipMemcpy(&h_partsum, d_partsum, num_blocks*sizeof(float), hipMemcpyDeviceToHost);

	for(int i=0; i<num_blocks; i++)
	{
//		printf("%.3f ", h_partsum[i]);
		total_sum += h_partsum[i];
	}
	std:: cout << "result = " << total_sum << "\n";
}

