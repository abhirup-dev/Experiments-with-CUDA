#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <vector>
#define num 25

__global__ void gpuAdd(int *d_a, int *d_b, int* d_c, int N=num)
{
	int tid = blockIdx.x;
	if(tid < N)
	{
		d_c[tid] = d_a[tid] + d_b[tid];
		printf("%d + %d = %d\n", d_a[tid], d_b[tid], d_c[tid]);
	}
}
void cpuAdd(int *h_a, int *h_b, int *h_c, int N=num)
{
	for(int i = 0; i < N; i++)
		h_c[i] = h_a[i] + h_b[i];
}
void cpuAdd_vec(std::vector<int> &h_a, std::vector<int> &h_b, std::vector<int> &h_c, int N=num)
{
	for(int i = 0; i < N; i++)
		h_c[i] = h_a[i] + h_b[i];
}
int main(void)
{
//	int N;
//	std::cout << "N?";
//	std::cin >> N;
	int N=num;
	std::cout << "N is " << num << "\n";
	int *d_a, *d_b, *d_c;//device pointer to store answer
	std::cout <<"Device allocate.. ";
	hipMalloc((void**)&d_a, N*sizeof(int));
	hipMalloc((void**)&d_b, N*sizeof(int));
	hipMalloc((void**)&d_c, N*sizeof(int));
	std::vector<int> h_a(N), h_b(N), h_c(N);
//	int
//		*h_a = (int*)malloc(N*sizeof(int)),
//		*h_b = (int*)malloc(N*sizeof(int)),
//		*h_c = (int*)malloc(N*sizeof(int));
	std::cout << "Allocated\n";
	for(int i=0; i<N; i++)
	{
		h_a[i] = i;
		h_b[i] = i * i;
		h_c[i] = i;
	}
	std::cout << "Finished!!!\n";
	//copy host to device
	hipMemcpy(d_a, h_a.data(), N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b.data(), N*sizeof(int), hipMemcpyHostToDevice);
	std::cout << "Ported to device\n";


	clock_t start, end;
//	start = clock();
////	cpuAdd(h_a, h_b, h_c);
//	cpuAdd_vec(h_a, h_b, h_c);
//	end = clock();
//	std:: cout << "CPU time: " << (double)(end-start)/ CLOCKS_PER_SEC << "\n";
	start = clock();
	gpuAdd <<<N, 1>>> (d_a, d_b, d_c, N);
	hipDeviceSynchronize();
	end = clock();
	std:: cout << "GPU time: " << (double)(end-start)/ CLOCKS_PER_SEC <<'\n';

//	free(h_a);
//	free(h_b);
//	free(h_c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
