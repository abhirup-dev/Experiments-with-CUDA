#include "stdio.h"
#include<iostream>
#include <hip/hip_runtime.h>


//Defining two constants
__constant__ float constant_f;
__constant__ float constant_g;
#define N 5

//Kernel function for using constant memory
__global__ void gpu_constant_memory(float *d_in, float *d_out)
{
  //Getting thread index for current kernel
  int tid = threadIdx.x;
  d_out[tid] = constant_f*d_in[tid] + constant_g;
}
int main()
{
	float h_x[N], h_y[N], h_f, h_g;
	float *d_x, *d_y;
	h_f = 5; h_g = 10;
	for(int i=0; i<N; i++)
		h_x[i] = i;
	hipMalloc(&d_x, N*sizeof(float));
	hipMalloc(&d_y, N*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(constant_f), &h_f, sizeof(float), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(constant_g), &h_g, sizeof(float));
	hipMemcpy(d_x, &h_x, N*sizeof(float), hipMemcpyHostToDevice);
	gpu_constant_memory<<<1,N>>>(d_x, d_y);

	hipMemcpy(&h_y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

	for(int i=0; i<N; i++)
		printf("%0.3f * %0.3f + %0.3f = %0.3f\n", h_x[i], h_f, h_g, h_y[i]);

	hipFree(d_x);
	hipFree(d_y);
}
