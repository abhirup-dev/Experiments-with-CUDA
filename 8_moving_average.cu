#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 10
__global__ void gpu_shared_mem(float *d)
{
	int i, idx = threadIdx.x;
	float avg, sum=0.0;

	//Defining shared memory
	__shared__ float sh_arr[N];
	sh_arr[idx] = d[idx];

	__syncthreads();

	for(i=0; i<=idx; i++)
		sum += sh_arr[i];
	avg = sum / (idx+1.0f);

	d[idx] = avg;
}
int main(void)
{
	float h[10], *d;
	for(int i=0; i<N; i++)
		h[i] = i;

	hipMalloc(&d, sizeof(float)*N);
	hipMemcpy(d, h, sizeof(float)*N, hipMemcpyHostToDevice);

	gpu_shared_mem<<<1, N>>>(d);

	hipMemcpy(h, d, sizeof(float)*N, hipMemcpyDeviceToHost);

	printf("Averaged array: ");
	for(int i=0; i<N; i++)
		printf("%f ", h[i]);
	printf("\n");

}
