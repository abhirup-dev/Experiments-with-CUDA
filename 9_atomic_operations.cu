#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define N 10
#define num_threads 10000

__global__ void increment_naive(int *d)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	tid = tid % N;
	d[tid] += 1;
}
__global__ void increment_atomic(int *d)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	tid = tid % N;
	atomicAdd(&d[tid], 1);
}
int main()
{
	int h[N], *d;
	hipMalloc(&d, sizeof(int)*N);
	hipMemset(d, 0, sizeof(int)*N);
	increment_naive<<<(num_threads/N), N>>>(d);
	hipMemcpy(h, d, sizeof(int)*N, hipMemcpyDeviceToHost);

	for(int i=0; i<N; i++)
		std::cout << h[i] << "\n";

	hipMemset(d, 0, sizeof(int)*N);
	increment_atomic<<<(num_threads/N), N>>>(d);
	hipMemcpy(h, d, sizeof(int)*N, hipMemcpyDeviceToHost);

	for(int i=0; i<N; i++)
		std::cout << h[i] << "\n";
}
//12
//12
//12
//12
//12
//12
//12
//12
//12
//12
//1000
//1000
//1000
//1000
//1000
//1000
//1000
//1000
//1000
//1000
