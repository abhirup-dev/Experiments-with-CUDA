#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>
#define num 10000000

__global__ void gpuSquare(int *d_a, int *d_b, int N=num)
{
	int tid = blockIdx.x;
	if(tid < N)
		d_b[tid] = d_a[tid] * d_a[tid];
}
void cpuSquare(std::vector<int> &h_a, std::vector<int> &h_b)
{
	for(int i=0; i<h_a.size(); i++)
		h_b[i] = h_a[i]*h_a[i];
}
int main(void)
{
//	int N;
//	std::cout << "N?";
//	std::cin >> N;
	int N=num;
	std::cout << "N is " << num << "\n";
	int *d_a, *d_b;//device pointer to store answer
	std::cout <<"Device allocate.. ";
	hipMalloc((void**)&d_a, N*sizeof(int));
	hipMalloc((void**)&d_b, N*sizeof(int));
	std::vector<int> h_a(N), h_b(N);
	std::cout << "Allocated\n";
	for(int i=0; i<N; i++)
	{
		h_a[i] = i;
	}
	std::cout << "Finished!!!\n";
	//copy host to device
	hipMemcpy(&d_a, h_a.data(), N*sizeof(int), hipMemcpyHostToDevice);
	std::cout << "Ported to device\n";
	clock_t start,end;
	start = clock();
	cpuSquare(h_a, h_b);
	end = clock();
	std:: cout << "CPU time: " << (double)(end-start)/ CLOCKS_PER_SEC << "\n";
	start = clock();
	gpuSquare <<<N, 1>>> (d_a, d_b);
//	hipDeviceSynchronize();
	hipDeviceSynchronize();
	end = clock();
	std:: cout << "GPU time: " << (double)(end-start)/ CLOCKS_PER_SEC <<'\n';

//	free(h_a);
//	free(h_b);
//	free(h_c);
	hipFree(d_a);
	hipFree(d_b);
	return 0;
}
