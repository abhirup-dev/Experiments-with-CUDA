#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>

#define N 10
#define num_threads 10
texture <float, 1, hipReadModeElementType> textureRef;

__global__ void gpu_texture_memory(int n, float *d_out)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < n)
	{
		float temp = tex1D(textureRef, float(idx));
		d_out[idx] = temp*temp;
	}
}
int main()
{
	int num_blocks = std::ceil(N / num_threads);
	float *d_out,
		*h_out = (float*)malloc(N*sizeof(float));
	hipMalloc(&d_out, sizeof(float)*N);
	float h_in[N];
	for(int i=0; i<N; i++)
		h_in[i] = float(i);

	//define CUDA array
	hipArray *cuArr;

	//set channel description of array (acc. to texture)
	hipMallocArray(&cuArr, &textureRef.channelDesc, N, 1);

	//copy to array
	hipMemcpyToArray(cuArr, 0, 0, h_in, sizeof(float)*N, hipMemcpyHostToDevice);

	//bind texture to array
	hipBindTextureToArray(textureRef, cuArr);

	gpu_texture_memory <<<num_blocks, num_threads>>>(N, d_out);

	hipMemcpy(h_out, d_out, sizeof(float)*N, hipMemcpyDeviceToHost);

	printf("Use of Texture memory on GPU: \n");
	  // Print the result
    for (int i = 0; i < N; i++)
    {
	  printf("Result at %f is : %f\n", h_in[i],h_out[i]);
    }
    free(h_out);
    hipFree(d_out);
    hipFreeArray(cuArr);
    hipUnbindTexture(textureRef);
}
