#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <memory>
__global__ void samplekernel(void)
{
	printf("Thread #[%d] in block#[%d]\n", threadIdx.x, blockIdx.x);
}

int main(void)
{
	int device;
	hipDeviceProp_t device_property;
	hipGetDevice(&device);
	printf("ID of device: %d\n", device);
	memset(&device_property, 0, sizeof(hipDeviceProp_t));
	device_property.major = 1;
	device_property.minor = 3;
	hipChooseDevice(&device, &device_property);
	printf("ID of device which supports double precision is: %d\n", device);
	hipSetDevice(device);
	samplekernel << < 2, 10>> >();
	hipDeviceSynchronize();
	std::cout << "Finished all threads\n";
	return 0;
}
