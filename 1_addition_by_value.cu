#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void gpuAdd(int d_a, int d_b, int* d_c)
{
	*d_c = d_a + d_b;
}
int main(void)
{
	int h_c;//host var to store answer
	int *d_c;//device pointer to store answer

	hipMalloc((void**) &d_c, sizeof(int));

	gpuAdd << <1, 1 >> > (3, 5, d_c);
	//copy results from device to host
	hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("3 + 5 = %d\n", h_c);

	hipFree(d_c);
	return 0;
}
