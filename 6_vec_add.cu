#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <algorithm>
#define num 25

__global__ void gpuAdd(int *d_a, int *d_b, int* d_c, int N=num)
{
	printf("%d -- ", threadIdx.x);
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	int sum=0;
	while(tid < N)
	{
		d_c[tid] = d_a[tid] + d_b[tid];
		printf("#[%d,%d] %d + %d = %d\n", tid,threadIdx.x, d_a[tid], d_b[tid], d_c[tid]);
		tid += blockDim.x * gridDim.x;
//		sum += d_c[tid];
	}
//	printf("#[%d] sum=%d\n", threadIdx.x, sum);
}
void cpuAdd(std::vector<int> &h_a, std::vector<int> &h_b, std::vector<int> &h_c, int N=num)
{
	for(int i = 0; i < N; i++)
		h_c[i] = h_a[i] + h_b[i];
}
int main(void)
{
	int N;
	std::cout << "N?";
	std::cin >> N;
//	int N=num;
	std::cout << "N is " << num << "\n";
	int *d_a, *d_b, *d_c;//device pointer to store answer
	std::cout <<"Device allocate.. ";
	hipMalloc((void**)&d_a, N*sizeof(int));
	hipMalloc((void**)&d_b, N*sizeof(int));
	hipMalloc((void**)&d_c, N*sizeof(int));

//	std::vector<int> h_a(N), h_b(N), h_c(N);
	int
		*h_a = (int*)malloc(N*sizeof(int)),
		*h_b = (int*)malloc(N*sizeof(int)),
		*h_c = (int*)malloc(N*sizeof(int));

	std::cout << "Allocated\n";
	for(int i=0; i<N; i++)
	{
		h_a[i] = i;
		h_b[i] = i * i;
	}
	std::cout << "Finished!!!\n";

	//copy host to device
	hipMemcpy(d_a, h_a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, N*sizeof(int), hipMemcpyHostToDevice);
	std::cout << "Ported to device\n";

	clock_t start, end;
	start = clock();
	gpuAdd <<<5, 7>>> (d_a, d_b, d_c, N);
	hipDeviceSynchronize();
	end = clock();
	std:: cout << "GPU time: " << (double)(end-start)/ CLOCKS_PER_SEC <<'\n';
//	cudaMemcpy(h_c, d_c, N*sizeof(int), cudaMemcpyDeviceToHost);

//	std::for_each(h_c, h_c+N, [](int x){
//		std::cout << x << "\n";
//	});

	free(h_a);
	free(h_b);
	free(h_c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
//N?25
//N is 1000000
//Device allocate.. Allocated
//Finished!!!
//Ported to device
//tid #[0] x #[0]
//tid #[1] x #[1]
//tid #[2] x #[2]
//tid #[3] x #[3]
//tid #[4] x #[4]
//tid #[10] x #[0]
//tid #[11] x #[1]
//tid #[12] x #[2]
//tid #[13] x #[3]
//tid #[14] x #[4]
//tid #[5] x #[0]
//tid #[6] x #[1]
//tid #[7] x #[2]
//tid #[8] x #[3]
//tid #[9] x #[4]
//tid #[15] x #[0]
//tid #[16] x #[1]
//tid #[17] x #[2]
//tid #[18] x #[3]
//tid #[19] x #[4]
//tid #[20] x #[0]
//tid #[21] x #[1]
//tid #[22] x #[2]
//tid #[23] x #[3]
//tid #[24] x #[4]
//GPU time: 0.000183
