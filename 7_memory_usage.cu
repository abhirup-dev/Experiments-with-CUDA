#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define N 5

__global__ void gpu_global_memory(int *d_a)
{
	d_a[threadIdx.x] = threadIdx.x;
}
__global__ void gpu_local_memory(int d_in)
{
	int t_local;
	t_local = d_in * threadIdx.x;
	printf("Val of local var in current thread is %d\n", t_local);
}
__global__ void gpu_shared_memory(float *d_a)
{
	int i, idx = threadIdx.x;
}
int main(void)
{
	int h_a[N]; int *d_a;

	//	writing in Global Memory
	hipMalloc(&d_a, N*sizeof(int));
	hipMemcpy(d_a, h_a, N*sizeof(int), hipMemcpyHostToDevice);
	gpu_global_memory<<<1,N>>>(d_a);
	hipDeviceSynchronize();
	hipMemcpy(h_a, d_a, N*sizeof(int), hipMemcpyDeviceToHost);
	printf("Array in Global Memory is: \n");
	for(int i=0; i<N; i++)
		printf("At Index: %d --> %d \n", i, h_a[i]);

	// writing in Local Memory
	printf("Use of Local memory on GPU.\n");
	gpu_local_memory <<<1,N>>>(5);
	hipDeviceSynchronize();
}
